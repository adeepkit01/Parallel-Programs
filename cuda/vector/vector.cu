
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
 
__global__ void vecAdd(double *a, double *b, double *c, int n)
{
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if (id < n)
        c[id] = a[id] + b[id];
}
 
int main( int argc, char* argv[] )
{
    int n = 100000;
 
    double *h_a;
    double *h_b;
    double *h_c;
 
    double *d_a;
    double *d_b;
    double *d_c;
 
    size_t bytes = n*sizeof(double);

    srand(time(NULL));
 
    h_a = (double*)malloc(bytes);
    h_b = (double*)malloc(bytes);
    h_c = (double*)malloc(bytes);
 
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);
 
    int i;
    for( i = 0; i < n; i++ ) {
        h_a[i] = rand ()%100;
        h_b[i] = rand ()%100;
    }

    float time;
    hipEvent_t start, stop;

    hipEventCreate(&start) ;
    hipEventCreate(&stop) ;
    hipEventRecord(start, 0) ;

    hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy( d_b, h_b, bytes, hipMemcpyHostToDevice);

    hipEventRecord(stop, 0) ;
    hipEventSynchronize(stop) ;
    hipEventElapsedTime(&time, start, stop) ;

    printf("w %f\n", time);
 
    int blockSize, gridSize;
    
    blockSize = atoi(argv[1]);
 
    gridSize = (int)ceil((float)n/blockSize);
 
    hipEventCreate(&start) ;
    hipEventCreate(&stop) ;
    hipEventRecord(start, 0) ;

    vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);

    hipEventRecord(stop, 0) ;
    hipEventSynchronize(stop) ;
    hipEventElapsedTime(&time, start, stop) ;

    printf("e %f\n", time);
 
    hipEventCreate(&start) ;
    hipEventCreate(&stop) ;
    hipEventRecord(start, 0) ;

    hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost );

    hipEventRecord(stop, 0) ;
    hipEventSynchronize(stop) ;
    hipEventElapsedTime(&time, start, stop) ;

    printf("w %f", time);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
 
    free(h_a);
    free(h_b);
    free(h_c);
 
    return 0;
}
